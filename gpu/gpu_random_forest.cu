#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <chrono>
#include <unistd.h>
#include <hip/hip_runtime.h>


/* different GPU kernel versions 

#define GPU_CSR
#define GPU_HIER
//if GPU_HIER is defined, then at least one of the following needs to be defined
#define ITER
#define ET2
#define ET3
#define ET4
#define ET5

*/

using namespace std;
using namespace std::chrono;
milliseconds running_time;

#define TIMING

#ifdef TIMING
  #define INIT_TIMER auto start = std::chrono::high_resolution_clock::now();
  #define START_TIMER  start = std::chrono::high_resolution_clock::now();
  #define STOP_TIMER(name) running_time = std::chrono::duration_cast<std::chrono::milliseconds>( \
                                          std::chrono::high_resolution_clock::now()-start); \
                           std::cout << "RUNTIME of " << name << ": " << running_time.count() << " ms " << std::endl; 
#else
  #define INIT_TIMER
  #define START_TIMER
  #define STOP_TIMER(name)
#endif

#ifdef GPU_HIER
__global__ void
hier_kernel_metadata_atomicFree(
  unsigned num_of_trees           ,
  unsigned *prefix_sum_subtree_nums        ,
  float    *nodes                          ,
  unsigned *idx_to_subtree                 ,  
  unsigned *leaf_idx_boundry               ,
  unsigned *subtree_has_leaf_arr           ,
  unsigned *g_subtree_nodes_offset         ,  
  unsigned *g_subtree_idx_to_subtree_offset,  

  unsigned num_of_queries         ,
  unsigned num_of_features        ,
  float *queries                  ,
  unsigned *results                  
){

    for (int tid = blockDim.x*blockIdx.x + threadIdx.x; tid < num_of_queries; tid += blockDim.x*gridDim.x){
      //fetch a new query
      float * row = queries + tid*num_of_features; 
         for(int tree_num = 0; tree_num < num_of_trees; ++tree_num){
            //go over trees
            unsigned tree_off_set = prefix_sum_subtree_nums[tree_num];
            unsigned curr_subtree_idx = 0 ;  

            //iterate over subtree
            while (true){
                //fetch the subtree nodes
                const float *subtree_node_list;
                subtree_node_list = nodes + g_subtree_nodes_offset[tree_off_set+curr_subtree_idx]*3 ;
        
                //fetch subtree_leaf_idx_boundry
                const unsigned subtree_leaf_idx_boundry = leaf_idx_boundry[tree_off_set+curr_subtree_idx];
                const unsigned subtree_has_leaf = subtree_has_leaf_arr[tree_off_set+curr_subtree_idx];
        
                //fetch subtree_idx_to_other_subtree
                const unsigned *subtree_idx_to_subtree = idx_to_subtree + g_subtree_idx_to_subtree_offset[tree_off_set+curr_subtree_idx]*2;
        
                //iterate over nodes in a subtree
                bool return_from_curr_tree = false;
                
                //start from node 0
                unsigned curr_node = 0;
                
                //start a recursive/iterative subtree traversal

        if (subtree_has_leaf)
        {
                while (true){
                    unsigned feature_id = subtree_node_list[curr_node*3];
                    float node_value    = subtree_node_list[curr_node*3+1];
                    unsigned is_tree_leaf    = subtree_node_list[curr_node*3+2];
                    // if node is leaf, then the prediction is over, we return the predicted value in node_value (in a tree leaf, node_value holds the predicted result)
                    if (is_tree_leaf==1)
                    { 
                      results[tid]+=(unsigned)node_value; 
                      return_from_curr_tree = true; 
                      goto SUBTREE_END; 
                    }
                    // if node is not leaf, we need two comparisons to decide if we keep traverse inside current subtree, or we go to another subtree
                    bool not_subtree_bottom = curr_node < subtree_leaf_idx_boundry;
                    bool go_left = row[feature_id] <= node_value;
                    // if not reach bottom of subtree, keep iterating using 2*i+1 or 2*i+2
                    if (not_subtree_bottom){
                        // go to left child in subtree
                        if (go_left)
                            curr_node = curr_node*2 + 1;
                        // go to right child in subtree
                        else
                            curr_node = curr_node*2 + 2;
                    // if reach bottom of subtree, then we need to go to another subtree
                    } else{
                        unsigned leaf_idx = curr_node - subtree_leaf_idx_boundry;
                        if (go_left)
                            curr_subtree_idx = subtree_idx_to_subtree[2*leaf_idx];
                        else
                            curr_subtree_idx = subtree_idx_to_subtree[2*leaf_idx+1];
                        //stop the iterating of the current subtree, jump to the outer loop
                        //break;
                        goto SUBTREE_END;
                    }
                }
        }
        else{
                while (true){
                    unsigned feature_id = subtree_node_list[curr_node*3];
                    float node_value    = subtree_node_list[curr_node*3+1];
                    //unsigned is_tree_leaf    = subtree_node_list[curr_node*3+2];

                    // if node is leaf, then the prediction is over, we return the predicted value in node_value (in a tree leaf, node_value holds the predicted result)
                    //if (is_tree_leaf==1){ atomicAdd(results+tid, (unsigned)node_value); return_from_curr_tree = true; break; }
                    //if (is_tree_leaf==1){ atomicAdd(results+tid, (unsigned)node_value); return_from_curr_tree = true; goto SUBTREE_END; }
                    // if node is not leaf, we need two comparisons to decide if we keep traverse inside current subtree, or we go to another subtree
                    bool not_subtree_bottom = curr_node < subtree_leaf_idx_boundry;
                    bool go_left = row[feature_id] <= node_value;
                    // if not reach bottom of subtree, keep iterating using 2*i+1 or 2*i+2
                    if (not_subtree_bottom){
                        // go to left child in subtree
                        if (go_left)
                            curr_node = curr_node*2 + 1;
                        // go to right child in subtree
                        else
                            curr_node = curr_node*2 + 2;
                    // if reach bottom of subtree, then we need to go to another subtree
                    } else{
                        unsigned leaf_idx = curr_node - subtree_leaf_idx_boundry;
                        if (go_left)
                            curr_subtree_idx = subtree_idx_to_subtree[2*leaf_idx];
                        else
                            curr_subtree_idx = subtree_idx_to_subtree[2*leaf_idx+1];
                        //stop the iterating of the current subtree, jump to the outer loop
                        //break;
                        goto SUBTREE_END;
                    }
                }

        }
                //end subtree
                //if return from curr tree, skip all rest of subtrees, break from looping over subtrees
SUBTREE_END:
                if (return_from_curr_tree) break;
            }
//}
 }}
}

__global__ void
hier_kernel_metadata_atomic(
  unsigned num_of_trees           ,
  unsigned *prefix_sum_subtree_nums        ,
  float    *nodes                          ,
  unsigned *idx_to_subtree                 ,  
  unsigned *leaf_idx_boundry               ,
  unsigned *subtree_has_leaf_arr           ,
  unsigned *g_subtree_nodes_offset         ,  
  unsigned *g_subtree_idx_to_subtree_offset,  

  unsigned num_of_queries         ,
  unsigned num_of_features        ,
  float *queries                  ,
  unsigned *results                  
){

   for(int tree_num=blockIdx.x; tree_num< num_of_trees; tree_num+=gridDim.x){
       //fetch a new tree
       unsigned tree_off_set = prefix_sum_subtree_nums[tree_num];
       for (int tid = threadIdx.x; tid < num_of_queries; tid += blockDim.x){
            //fetch a new query
            float *row = queries + tid*num_of_features; 
        
            unsigned curr_subtree_idx = 0 ;  

            //iterate over subtree
            while (true){
                //fetch the subtree nodes
                const float *subtree_node_list;
                subtree_node_list = nodes + g_subtree_nodes_offset[tree_off_set+curr_subtree_idx]*3 ;
        
                //fetch subtree_leaf_idx_boundry
                const unsigned subtree_leaf_idx_boundry = leaf_idx_boundry[tree_off_set+curr_subtree_idx];
                const unsigned subtree_has_leaf = subtree_has_leaf_arr[tree_off_set+curr_subtree_idx];
        
                //fetch subtree_idx_to_other_subtree
                const unsigned *subtree_idx_to_subtree = idx_to_subtree + g_subtree_idx_to_subtree_offset[tree_off_set+curr_subtree_idx]*2;
        
                //iterate over nodes in a subtree
                bool return_from_curr_tree = false;
                
                //start from node 0
                unsigned curr_node = 0;
                
                //start a recursive/iterative subtree traversal

        if (subtree_has_leaf)
        {
                while (true){
                    unsigned feature_id = subtree_node_list[curr_node*3];
                    float node_value    = subtree_node_list[curr_node*3+1];
                    unsigned is_tree_leaf    = subtree_node_list[curr_node*3+2];
                    // if node is leaf, then the prediction is over, we return the predicted value in node_value (in a tree leaf, node_value holds the predicted result)
                    if (is_tree_leaf==1)
                    { 
                      atomicAdd(results+tid, (unsigned)node_value); 
                      return_from_curr_tree = true; 
                      goto SUBTREE_END; 
                    }
                    // if node is not leaf, we need two comparisons to decide if we keep traverse inside current subtree, or we go to another subtree
                    bool not_subtree_bottom = curr_node < subtree_leaf_idx_boundry;
                    bool go_left = row[feature_id] <= node_value;
                    // if not reach bottom of subtree, keep iterating using 2*i+1 or 2*i+2
                    if (not_subtree_bottom){
                        // go to left child in subtree
                        if (go_left)
                            curr_node = curr_node*2 + 1;
                        // go to right child in subtree
                        else
                            curr_node = curr_node*2 + 2;
                    // if reach bottom of subtree, then we need to go to another subtree
                    } else{
                        unsigned leaf_idx = curr_node - subtree_leaf_idx_boundry;
                        if (go_left)
                            curr_subtree_idx = subtree_idx_to_subtree[2*leaf_idx];
                        else
                            curr_subtree_idx = subtree_idx_to_subtree[2*leaf_idx+1];
                        //stop the iterating of the current subtree, jump to the outer loop
                        //break;
                        goto SUBTREE_END;
                    }
                }
        }
        else{
                while (true){
                    unsigned feature_id = subtree_node_list[curr_node*3];
                    float node_value    = subtree_node_list[curr_node*3+1];
                    //unsigned is_tree_leaf    = subtree_node_list[curr_node*3+2];

                    // if node is leaf, then the prediction is over, we return the predicted value in node_value (in a tree leaf, node_value holds the predicted result)
                    //if (is_tree_leaf==1){ atomicAdd(results+tid, (unsigned)node_value); return_from_curr_tree = true; break; }
                    //if (is_tree_leaf==1){ atomicAdd(results+tid, (unsigned)node_value); return_from_curr_tree = true; goto SUBTREE_END; }
                    // if node is not leaf, we need two comparisons to decide if we keep traverse inside current subtree, or we go to another subtree
                    bool not_subtree_bottom = curr_node < subtree_leaf_idx_boundry;
                    bool go_left = row[feature_id] <= node_value;
                    // if not reach bottom of subtree, keep iterating using 2*i+1 or 2*i+2
                    if (not_subtree_bottom){
                        // go to left child in subtree
                        if (go_left)
                            curr_node = curr_node*2 + 1;
                        // go to right child in subtree
                        else
                            curr_node = curr_node*2 + 2;
                    // if reach bottom of subtree, then we need to go to another subtree
                    } else{
                        unsigned leaf_idx = curr_node - subtree_leaf_idx_boundry;
                        if (go_left)
                            curr_subtree_idx = subtree_idx_to_subtree[2*leaf_idx];
                        else
                            curr_subtree_idx = subtree_idx_to_subtree[2*leaf_idx+1];
                        //stop the iterating of the current subtree, jump to the outer loop
                        //break;
                        goto SUBTREE_END;
                    }
                }

        }
                //end subtree
                //if return from curr tree, skip all rest of subtrees, break from looping over subtrees
SUBTREE_END:
                if (return_from_curr_tree) break;
            }
//}
 }}
}
#endif //end GPU_HIER

#ifdef GPU_CSR 
__global__ void
csr_kernel(
  unsigned num_of_trees           ,
  unsigned *   node_list_idx      ,
  unsigned *   edge_list_idx      ,
  unsigned *   node_is_leaf_idx   ,
  unsigned *   node_features_idx  ,
  unsigned *   node_values_idx    ,

  unsigned *   node_list_total    ,
  unsigned *   edge_list_total    ,
  unsigned *   node_is_leaf_total ,
  unsigned *   node_features_total,
  float    *   node_values_total  ,

  unsigned num_of_queries         ,
  unsigned num_of_features        ,
  float *queries                  ,
  unsigned *results                  
){
    for (int tid = blockDim.x*blockIdx.x + threadIdx.x; tid < num_of_queries; tid += blockDim.x*gridDim.x){
            //fetch a new query
            float * row = queries + tid*num_of_features; 
            //go over trees
            for(int i=0; i< num_of_trees; ++i){
                //csr layout
                //unsigned num_of_nodes = node_list_idx[i+1]-node_list_idx[i]-1;
                unsigned * node_list = node_list_total + node_list_idx[i];
                unsigned * edge_list = edge_list_total + edge_list_idx[i];
                unsigned * node_is_leaf = node_is_leaf_total + node_is_leaf_idx[i];
                unsigned * node_features = node_features_total + node_features_idx[i];
                float * node_values = node_values_total + node_values_idx[i];
        
                //start from node 0
                unsigned curr_node = 0;
                //iterate over nodes in a subtree
                while (true){
                    unsigned feature_id    = node_features[curr_node]; 
                    float node_value       = node_values[curr_node];
                    unsigned is_tree_leaf  = node_is_leaf[curr_node];
                    // if node is leaf, then the prediction is over, we return the predicted value in node_value (in a tree leaf, node_value holds the predicted result)
                    if (is_tree_leaf==1){
                      //results[tid] = node_value;
//                        if (node_value == 1.0f){
                          atomicAdd(results+tid,(unsigned)node_value);
//                        }
                      break;
                    }
                    // if node is not leaf, we need two comparisons to decide if we keep traverse 
                    bool go_left = row[feature_id] <= node_value;
                    if (go_left)
                        curr_node = edge_list[node_list[curr_node]]; 
                    // go to right child in subtree
                    else
                        curr_node = edge_list[node_list[curr_node]+1]; 
                }
            }
    }     
}
#endif

__global__ void 
generate_results(unsigned num_of_queries, unsigned num_of_trees, unsigned * results){
    unsigned threshold = num_of_trees/2;
    for (int tid = blockDim.x*blockIdx.x + threadIdx.x; tid < num_of_queries; tid += blockDim.x*gridDim.x){
      if (results[tid] > threshold)
        results[tid] = 1;
      else
        results[tid] = 0;
    }
}


template <typename T>
unsigned read_arr(ifstream &infile, vector<T> &output ,string var_name);
template <typename T>
void read_2darr(ifstream &infile, vector<T> &output ,string var_name, unsigned &row, unsigned &cow);

float predict_tree_csr_layout(unsigned *node_list, unsigned *edge_list, unsigned *node_is_leaf, unsigned *node_features, float *node_values, float *row);

float predict_tree_gpu_layout(int num_of_trees, const unsigned *prefix_sum_subtree_nums, const float *nodes, const unsigned *idx_to_subtree, const unsigned *leaf_idx_boundry , const unsigned *g_subtree_nodes_offset, const unsigned *g_subtree_idx_to_subtree_offset, unsigned tree_num, float *row); 

int main(int argc, char **argv){

  //common data used by both csr and hier versions of GPU kernels
  ifstream infile;
  unsigned num_of_trees;
  unsigned tree_depth = atoi(argv[1]);
  INIT_TIMER 
  vector<unsigned> h_results;
  unsigned wrong_num = 0;
  dim3 gridD(60);
  dim3 blockD(256);
  hipSetDevice(0);

#ifdef GPU_HIER
  //read HIER data
  FILE *fHier = fopen("hier.csv","a+");

  if(!fHier){
    printf("Error open hier.csv file");
    exit(1);
  }
  fseek(fHier,0,SEEK_END);
  if (ftell(fHier) == 0){
    fprintf(fHier,"%-10s,%-20s,%-10s,%-10s,%-20s,%-20s\n","tree_depth","num_of_trees","mapping","st_depth","method","time");
  }

  //read HIER data
  infile.open("treefile_hier.txt");
  string str;
  char   c;
  infile >> str;
  if (str!=string("num_of_trees")) {
    cout << str << "error reading num_of_trees";
  }
  infile >> num_of_trees >> c;
  cout << str << "\n" << num_of_trees << "\n";
  infile >> str;
  if (str!=string("prefix_sum_subtree_nums")) {
    cout << str << "error reading prefix_sum_subtree_nums";
  }
  unsigned len_prefix_sum_subtree_nums;
  infile >> len_prefix_sum_subtree_nums >> c;
  cout << str << "\n" << len_prefix_sum_subtree_nums << "\n";
  vector<unsigned> prefix_sum_subtree_nums(len_prefix_sum_subtree_nums,0);
  for (unsigned i = 0; i<len_prefix_sum_subtree_nums; ++i){
    infile >> prefix_sum_subtree_nums[i] >> c;
  }
  infile >> str;
  if (str!=string("nodes")) {
    cout << str << "error reading nodes";
  }
  unsigned len_nodes;
  infile >> len_nodes >> c;
  cout << str << "\n" << len_nodes << "\n";
  vector<float> nodes(len_nodes);
  for (unsigned i=0;i< len_nodes;++i){
    infile >> nodes[i] >> c;
  }
  infile >> str;
  if (str!=string("idx_to_subtree")) {
    cout << str << "error reading idx_to_subtree";
  }
  unsigned len_idx_to_subtree;
  infile >> len_idx_to_subtree >> c;
  cout << str << "\n" << len_idx_to_subtree << "\n";
  vector<unsigned> idx_to_subtree(len_idx_to_subtree,0);
  for (unsigned i = 0; i<len_idx_to_subtree; ++i){
    infile >> idx_to_subtree[i] >> c;
  }
  infile >> str;
  if (str!=string("leaf_idx_boundry")) {
    cout << str << "error reading leaf_idx_boundry";
  }
  unsigned len_leaf_idx_boundry;
  infile >> len_leaf_idx_boundry >> c;
  cout << str << "\n" << len_leaf_idx_boundry << "\n";
  vector<unsigned> leaf_idx_boundry(len_leaf_idx_boundry,0);
  for (unsigned i = 0; i<len_leaf_idx_boundry; ++i){
    infile >> leaf_idx_boundry[i] >> c;
  }

  infile >> str;
  if (str!=string("subtree_has_leaf")) {
    cout << str << "error reading subtree_has_leaf";
  }
  unsigned len_subtree_has_leaf;
  infile >> len_subtree_has_leaf >> c;
  cout << str << "\n" << len_subtree_has_leaf << "\n";
  vector<unsigned> subtree_has_leaf(len_subtree_has_leaf,0);
  for (unsigned i = 0; i<len_subtree_has_leaf; ++i){
    infile >> subtree_has_leaf[i] >> c;
  }

  infile >> str;
  if (str!=string("g_subtree_nodes_offset")) {
    cout << str << "error reading g_subtree_nodes_offset";
  }
  unsigned len_g_subtree_nodes_offset;
  infile >> len_g_subtree_nodes_offset >> c;
  cout << str << "\n" << len_g_subtree_nodes_offset << "\n";
  vector<unsigned> g_subtree_nodes_offset(len_g_subtree_nodes_offset,0);
  for (unsigned i = 0; i<len_g_subtree_nodes_offset; ++i){
    infile >> g_subtree_nodes_offset[i] >> c;
  }
  infile >> str;
  if (str!=string("g_subtree_idx_to_subtree_offset")) {
    cout << str << "error reading g_subtree_idx_to_subtree_offset";
  }
  unsigned len_g_subtree_idx_to_subtree_offset;
  infile >> len_g_subtree_idx_to_subtree_offset >> c;
  cout << str << "\n" << len_g_subtree_idx_to_subtree_offset << "\n";
  vector<unsigned> g_subtree_idx_to_subtree_offset(len_g_subtree_idx_to_subtree_offset,0);
  for (unsigned i = 0; i<len_g_subtree_idx_to_subtree_offset; ++i){
    float tmp;
    infile >> tmp >> c;
    g_subtree_idx_to_subtree_offset[i] = tmp; 
  }
  infile.close();
#endif

#ifdef GPU_CSR
  //read CSR data
  infile.open("treefile_csr.txt");
  vector<unsigned>   node_list_idx      ; 
  vector<unsigned>   edge_list_idx      ;
  vector<unsigned>   node_is_leaf_idx   ;
  vector<unsigned>   node_features_idx  ;
  vector<unsigned>   node_values_idx    ;
  vector<unsigned>   node_list_total    ;
  vector<unsigned>   edge_list_total    ;
  vector<unsigned>   node_is_leaf_total ;
  vector<unsigned>   node_features_total;
  vector<float>   node_values_total  ;
  num_of_trees = read_arr(infile, node_list_idx          , "node_list_idx"         );
  //read_arr returns size of array being read, size of node_list_idx = num_of_trees + 1
  num_of_trees = num_of_trees - 1;
  read_arr(infile, edge_list_idx          , "edge_list_idx"         );
  read_arr(infile, node_is_leaf_idx       , "node_is_leaf_idx"      );
  read_arr(infile, node_features_idx      , "node_features_idx"     );
  read_arr(infile, node_values_idx        , "node_values_idx"       );
  read_arr(infile, node_list_total        , "node_list_total"       );
  read_arr(infile, edge_list_total        , "edge_list_total"       );
  read_arr(infile, node_is_leaf_total     , "node_is_leaf_total"    );
  read_arr(infile, node_features_total    , "node_features_total"   );
  read_arr(infile, node_values_total      , "node_values_total"     );
  infile.close();
#endif
 

 //read input data 
  infile.open("./tree_input.txt");
  vector<float> X_test;
  vector<float> y_test;
  unsigned row,col;
  read_2darr(infile, X_test, "X_test", row, col);
  cout << "X_test" << " with " << row << " rows" << " and " << col << " cols.\n";
  read_arr(infile, y_test, "y_test");
  infile.close();

//NOW we copy input and allocate output to/on GPU
  float *d_queries;
  hipMalloc((void**)&d_queries, sizeof(float)*row*col);

  unsigned *d_results;
  hipMalloc((void**)&d_results, sizeof(unsigned)*row);

  hipMemcpy(d_queries, X_test.data(), sizeof(float)*row*col, hipMemcpyHostToDevice);

#ifdef GPU_CSR
  cout << "Allocating csr data on GPU" << endl;

//NOW we have and need to copy these consolidated csr format to GPU 
  cout << "Copying csr data to GPU" << endl;
  // unsigned num_of_trees;
  //vector<unsigned>   node_list_idx      ; 
  //vector<unsigned>   edge_list_idx      ;
  //vector<unsigned>   node_is_leaf_idx   ;
  //vector<unsigned>   node_features_idx  ;
  //vector<unsigned>   node_values_idx    ;

  //vector<unsigned>   node_list_total    ;
  //vector<unsigned>   edge_list_total    ;
  //vector<unsigned>   node_is_leaf_total ;
  //vector<unsigned>   node_features_total;
  //vector<float>   node_values_total  ;

  unsigned *   d_node_list_idx      ; 
  unsigned *   d_edge_list_idx      ;
  unsigned *   d_node_is_leaf_idx   ;
  unsigned *   d_node_features_idx  ;
  unsigned *   d_node_values_idx    ;

  unsigned *   d_node_list_total    ;
  unsigned *   d_edge_list_total    ;
  unsigned *   d_node_is_leaf_total ;
  unsigned *   d_node_features_total;
  float    *   d_node_values_total  ;


  cudaMalloc((void **) &d_node_list_idx, sizeof(unsigned)*node_list_idx.size());
  cudaMalloc((void **) &d_edge_list_idx, sizeof(unsigned)*edge_list_idx.size());
  cudaMalloc((void **) &d_node_is_leaf_idx, sizeof(unsigned)*node_is_leaf_idx.size());
  cudaMalloc((void **) &d_node_features_idx, sizeof(unsigned)*node_features_idx.size());
  cudaMalloc((void **) &d_node_values_idx, sizeof(unsigned)*node_values_idx.size());
  
  cudaMalloc((void **) &d_node_list_total, sizeof(unsigned)*node_list_total.size());
  cudaMalloc((void **) &d_edge_list_total, sizeof(unsigned)*edge_list_total.size());
  cudaMalloc((void **) &d_node_is_leaf_total, sizeof(unsigned)*node_is_leaf_total.size());
  cudaMalloc((void **) &d_node_features_total, sizeof(unsigned)*node_features_total.size());
  cudaMalloc((void **) &d_node_values_total, sizeof(unsigned)*node_values_total.size());

  cudaMemcpy(d_node_list_idx,              node_list_idx.data()    ,         sizeof(unsigned)*node_list_idx.size(),cudaMemcpyHostToDevice );
  cudaMemcpy(d_edge_list_idx,              edge_list_idx.data()    ,         sizeof(unsigned)*edge_list_idx.size(),cudaMemcpyHostToDevice );
  cudaMemcpy(d_node_is_leaf_idx,        node_is_leaf_idx.data()    ,      sizeof(unsigned)*node_is_leaf_idx.size(),cudaMemcpyHostToDevice );
  cudaMemcpy(d_node_features_idx,      node_features_idx.data()    ,     sizeof(unsigned)*node_features_idx.size(),cudaMemcpyHostToDevice );
  cudaMemcpy(d_node_values_idx,          node_values_idx.data()    ,       sizeof(unsigned)*node_values_idx.size(),cudaMemcpyHostToDevice );

  cudaMemcpy(d_node_list_total,          node_list_total.data()    ,       sizeof(unsigned)*node_list_total.size(),cudaMemcpyHostToDevice );
  cudaMemcpy(d_edge_list_total,          edge_list_total.data()    ,       sizeof(unsigned)*edge_list_total.size(),cudaMemcpyHostToDevice );
  cudaMemcpy(d_node_is_leaf_total,    node_is_leaf_total.data()    ,    sizeof(unsigned)*node_is_leaf_total.size(),cudaMemcpyHostToDevice );
  cudaMemcpy(d_node_features_total,  node_features_total.data()    ,   sizeof(unsigned)*node_features_total.size(),cudaMemcpyHostToDevice );
  cudaMemcpy(d_node_values_total,      node_values_total.data()    ,     sizeof(unsigned)*node_values_total.size(),cudaMemcpyHostToDevice );


  cout << "Start executing csr format on GPU" << endl;
  cudaMemset(d_results, 0 , row*sizeof(unsigned));
  cout << cudaGetErrorName(cudaGetLastError()) << endl;
  START_TIMER
  csr_kernel<<<60,256>>>(
                          num_of_trees           ,
                          d_node_list_idx      ,
                          d_edge_list_idx      ,
                          d_node_is_leaf_idx   ,
                          d_node_features_idx  ,
                          d_node_values_idx    ,

                          d_node_list_total    ,
                          d_edge_list_total    ,
                          d_node_is_leaf_total ,
                          d_node_features_total,
                          d_node_values_total  ,

                          row                  ,
                          col                  ,
                          d_queries            ,
                          d_results                  
  );
  generate_results<<<60,256>>>(row, num_of_trees, d_results);
  cudaDeviceSynchronize();
  STOP_TIMER("csr kernel")
  cout << "Kernel returned:" << cudaGetErrorName(cudaGetLastError()) << endl;
  h_results.resize(row);
  cudaMemcpy( h_results.data(), d_results, sizeof(unsigned)*row, cudaMemcpyDeviceToHost);
  cudaDeviceSynchronize();

  wrong_num = 0;
  for(auto i=0; i < row;++i){
    if (h_results[i]!=y_test[i]){
      wrong_num++;
    }
  }
  cout << "csr result is wrong with this many: " << wrong_num << endl;
  cout << "accuracy rate: " << (float)(row-wrong_num)/(float)row << endl;

  //destroy csr related data on GPU
  cudaFree(d_node_list_idx)      ;             
  cudaFree(d_edge_list_idx)      ;    
  cudaFree(d_node_is_leaf_idx)   ;    
  cudaFree(d_node_features_idx)  ;   
  cudaFree(d_node_values_idx)    ;   
  cudaFree(d_node_list_total)    ;    
  cudaFree(d_edge_list_total)    ;   
  cudaFree(d_node_is_leaf_total) ;       
  cudaFree(d_node_features_total);   
  cudaFree(d_node_values_total)  ; 
#endif
  
#ifdef GPU_HIER
/*
//validate GPU hierarchcal layout on CPU
  cout << "Test csr and hier results with " << row << " test samples.\n";
  for(int query = 0; query < row; ++query){
    float* row = X_test.data() + query*col;
    for(int i = 0; i < num_of_trees; ++i){
      //hier layout
      float res1 = predict_tree_gpu_layout(num_of_trees, prefix_sum_subtree_nums.data(), nodes.data(), idx_to_subtree.data(), leaf_idx_boundry.data() ,g_subtree_nodes_offset.data(), g_subtree_idx_to_subtree_offset.data(), i , row); 
      //cout << "Result1:" << res1 << "\n";

      //csr layout
      unsigned num_of_nodes = node_list_idx[i+1]-node_list_idx[i]-1;
      unsigned * node_list = node_list_total.data() + node_list_idx[i];
      unsigned * edge_list = edge_list_total.data() + edge_list_idx[i];
      unsigned * node_is_leaf = node_is_leaf_total.data() + node_is_leaf_idx[i];
      unsigned * node_features = node_features_total.data() + node_features_idx[i];
      float * node_values = node_values_total.data() + node_values_idx[i];
      
      float res2 = predict_tree_csr_layout(node_list, edge_list, node_is_leaf, node_features, node_values, row);
      //cout << "Result2:" << res2 << "\n";
      if (res1!=res2){
        cout << "csr and hier results don't match" << res1 << " res1|res2 "<< res2 << "\n";
      }
    }
  }
  cout << "csr and hier results match, verification passes" << endl;
*/


//NOW we copy these hier tree format to GPU 
//       unsigned num_of_trees                              ;
//       vector<unsigned> prefix_sum_subtree_nums           ;
//       vector<float   > nodes                             ;
//       vector<unsigned> idx_to_subtree                    ;
//       vector<unsigned> leaf_idx_boundry                  ;
//       vector<unsigned> g_subtree_nodes_offset            ;
//       vector<unsigned> g_subtree_idx_to_subtree_offset   ;

  cout << "Allocating hier format on GPU" << endl;
  unsigned *d_prefix_sum_subtree_nums        ;   
  float    *d_nodes                          ;
  unsigned *d_idx_to_subtree                 ;
  unsigned *d_leaf_idx_boundry               ;
  unsigned *d_subtree_has_leaf               ;
  unsigned *d_g_subtree_nodes_offset         ;
  unsigned *d_g_subtree_idx_to_subtree_offset;
  
  cudaMalloc((void**)&d_prefix_sum_subtree_nums              ,sizeof( unsigned )*prefix_sum_subtree_nums.size()            );
  cudaMalloc((void**)&d_nodes                                ,sizeof( float    )*nodes.size()                              );
  cudaMalloc((void**)&d_idx_to_subtree                       ,sizeof( unsigned )*idx_to_subtree.size()                     );
  cudaMalloc((void**)&d_leaf_idx_boundry                     ,sizeof( unsigned )*leaf_idx_boundry.size()                   );
  cudaMalloc((void**)&d_subtree_has_leaf                     ,sizeof( unsigned )*subtree_has_leaf.size()                   );
  cudaMalloc((void**)&d_g_subtree_nodes_offset               ,sizeof( unsigned )*g_subtree_nodes_offset.size()             );
  cudaMalloc((void**)&d_g_subtree_idx_to_subtree_offset      ,sizeof( unsigned )*g_subtree_idx_to_subtree_offset.size()    );
  
  cout << "Copying hier format to GPU" << endl;
  cudaMemcpy( d_prefix_sum_subtree_nums        ,prefix_sum_subtree_nums.data()        ,sizeof( unsigned )*prefix_sum_subtree_nums.size()        ,cudaMemcpyHostToDevice);
  cudaMemcpy( d_nodes                          ,nodes.data()                          ,sizeof( float    )*nodes.size()                          ,cudaMemcpyHostToDevice);
  cudaMemcpy( d_idx_to_subtree                 ,idx_to_subtree.data()                 ,sizeof( unsigned )*idx_to_subtree.size()                 ,cudaMemcpyHostToDevice);
  cudaMemcpy( d_leaf_idx_boundry               ,leaf_idx_boundry.data()               ,sizeof( unsigned )*leaf_idx_boundry.size()               ,cudaMemcpyHostToDevice);
  cudaMemcpy( d_subtree_has_leaf               ,subtree_has_leaf.data()               ,sizeof( unsigned )*subtree_has_leaf.size()               ,cudaMemcpyHostToDevice);
  cudaMemcpy( d_g_subtree_nodes_offset         ,g_subtree_nodes_offset.data()         ,sizeof( unsigned )*g_subtree_nodes_offset.size()         ,cudaMemcpyHostToDevice);
  cudaMemcpy( d_g_subtree_idx_to_subtree_offset,g_subtree_idx_to_subtree_offset.data(),sizeof( unsigned )*g_subtree_idx_to_subtree_offset.size(),cudaMemcpyHostToDevice);
  
  cout << "Start executing hier atomic free kernel on GPU" << endl;
  cout << cudaGetErrorName(cudaGetLastError()) << endl;
  //reset result array to 0
  cudaMemset(d_results, 0 , row*sizeof(unsigned));
  START_TIMER
  hier_kernel_metadata_atomicFree<<<gridD,blockD>>>(
                          num_of_trees                     ,
                          d_prefix_sum_subtree_nums        ,
                          d_nodes                          ,
                          d_idx_to_subtree                 ,  
                          d_leaf_idx_boundry               ,
                          d_subtree_has_leaf               ,
                          d_g_subtree_nodes_offset         ,  
                          d_g_subtree_idx_to_subtree_offset,  
  
                          row                              ,
                          col                              ,
                          d_queries                        ,
                          d_results                  
  );
  generate_results<<<gridD,blockD>>>(row, num_of_trees, d_results);
  cudaDeviceSynchronize();
  STOP_TIMER("hier kernel")
  cout << "Kernel returned:" << cudaGetErrorName(cudaGetLastError()) << endl;

  h_results.resize(row);
  cudaMemcpy( h_results.data(), d_results, sizeof(unsigned)*row, cudaMemcpyDeviceToHost);
  cudaDeviceSynchronize();

  wrong_num = 0;
  for(auto i=0; i < row;++i){
    if (h_results[i]!=y_test[i]){
      wrong_num++;
    }
  }
  cout << "hier result is wrong with this many: " << wrong_num << endl;
  cout << "accuracy rate: " << (float)(row-wrong_num)/(float)row << endl;
  fprintf(fHier,"%-10d,%-20d,%-10s,%-10d,%-20s,%-20ld\n",tree_depth,num_of_trees,"atomicFree",5,"iter_thread_meta",running_time);



  cout << "Start executing hier atomic kernel on GPU" << endl;
  //reset result array to 0
  cudaMemset(d_results, 0 , row*sizeof(unsigned));
  START_TIMER
  hier_kernel_metadata_atomic<<<gridD,blockD>>>(
                          num_of_trees                     ,
                          d_prefix_sum_subtree_nums        ,
                          d_nodes                          ,
                          d_idx_to_subtree                 ,  
                          d_leaf_idx_boundry               ,
                          d_subtree_has_leaf               ,
                          d_g_subtree_nodes_offset         ,  
                          d_g_subtree_idx_to_subtree_offset,  
  
                          row                              ,
                          col                              ,
                          d_queries                        ,
                          d_results                  
  );
  generate_results<<<gridD,blockD>>>(row, num_of_trees, d_results);
  cudaDeviceSynchronize();
  STOP_TIMER("hier kernel")
  cout << "Kernel returned:" << cudaGetErrorName(cudaGetLastError()) << endl;

  h_results.resize(row);
  cudaMemcpy( h_results.data(), d_results, sizeof(unsigned)*row, cudaMemcpyDeviceToHost);
  cudaDeviceSynchronize();

  wrong_num = 0;
  for(auto i=0; i < row;++i){
    if (h_results[i]!=y_test[i]){
      wrong_num++;
    }
  }
  cout << "hier result is wrong with this many: " << wrong_num << endl;
  cout << "accuracy rate: " << (float)(row-wrong_num)/(float)row << endl;
  fprintf(fHier,"%-10d,%-20d,%-10s,%-10d,%-20s,%-20ld\n",tree_depth,num_of_trees,"atomic",5,"iter_thread_meta",running_time);

  fclose(fHier);
#endif

  //main returns
  return 0;
}

//predict the result over a decision_tree
float predict_tree_gpu_layout(int num_of_trees, const unsigned *prefix_sum_subtree_nums, const float *nodes, const unsigned *idx_to_subtree, const unsigned *leaf_idx_boundry , const unsigned *g_subtree_nodes_offset, const unsigned *g_subtree_idx_to_subtree_offset, unsigned tree_num, float *row) {

    unsigned tree_off_set = prefix_sum_subtree_nums[tree_num];
    //unsigned num_of_subtrees = prefix_sum_subtree_nums[tree_num+1] - tree_off_set;

    unsigned curr_subtree_idx = 0 ;  
    //iterate over subtree
    while (true){
        //fetch the subtree nodes
        const float *subtree_node_list;
        subtree_node_list = nodes + g_subtree_nodes_offset[tree_off_set+curr_subtree_idx]*3 ;

        //fetch subtree_leaf_idx_boundry
        const unsigned subtree_leaf_idx_boundry = leaf_idx_boundry[tree_off_set+curr_subtree_idx];

        //fetch subtree_idx_to_other_subtree
        const unsigned *subtree_idx_to_subtree = idx_to_subtree + g_subtree_idx_to_subtree_offset[tree_off_set+curr_subtree_idx]*2;

        //start from node 0
        unsigned curr_node = 0;
        //iterate over nodes in a subtree
        while (true){
            unsigned feature_id = subtree_node_list[curr_node*3];
            float node_value    = subtree_node_list[curr_node*3+1];
            unsigned is_tree_leaf    = subtree_node_list[curr_node*3+2];

            // if node is leaf, then the prediction is over, we return the predicted value in node_value (in a tree leaf, node_value holds the predicted result)
            if (is_tree_leaf==1)
                return node_value;

            // if node is not leaf, we need two comparisons to decide if we keep traverse inside current subtree, or we go to another subtree
            bool not_subtree_bottom = curr_node < subtree_leaf_idx_boundry;
            bool go_left = row[feature_id] <= node_value;

            // if not reach bottom of subtree, keep iterating using 2*i+1 or 2*i+2
            if (not_subtree_bottom){
                // go to left child in subtree
                if (go_left)
                    curr_node = curr_node*2 + 1;
                // go to right child in subtree
                else
                    curr_node = curr_node*2 + 2;
            // if reach bottom of subtree, then we need to go to another subtree
            } else{
                unsigned leaf_idx = curr_node - subtree_leaf_idx_boundry;
                if (go_left)
                    curr_subtree_idx = subtree_idx_to_subtree[2*leaf_idx];
                else
                    curr_subtree_idx = subtree_idx_to_subtree[2*leaf_idx+1];
                //stop the iterating of the current subtree, jump to the outer loop
                break;
            }
        }
    }
}

template <typename T>
unsigned read_arr(ifstream &infile, vector<T> &output, string var_name){
  string str;
  char c;
  infile >> str;
  if (str!=var_name) {
    cout << str << "error reading " << var_name << endl;
  }
  unsigned len;
  infile >> len >> c;
  output.resize(len);
  for (unsigned i = 0; i<len; ++i){
    infile >> output[i] >> c;
  }
//  cout << "Read " << str << " with " << len << " elements\n";
  return len;
}

template <typename T>
void read_2darr(ifstream &infile, vector<T> &output, string var_name, unsigned &row, unsigned &col){
  string str;
  char c;
  infile >> str;
  if (str!=var_name) {
    cout << str << "error reading " << var_name << endl;
  }
  unsigned nrow,ncol;
  infile >> nrow >> c >> ncol >> c;
  row = nrow;
  col = ncol;
  output.resize(nrow*ncol);
  for (unsigned i = 0; i<nrow*ncol; ++i){
    infile >> output[i] >> c;
  }
//  cout << "Read " << str << " with " << nrow << " rows" << " and " << ncol << " cols.\n";
}

//    node_list = tree[1]
//    edge_list = tree[2]
//    node_is_leaf = tree[3]
//    node_features = tree[4]
//    node_values = tree[5]
float predict_tree_csr_layout(unsigned *node_list, unsigned *edge_list, unsigned *node_is_leaf, unsigned *node_features, float *node_values, float *row){
    //start from node 0
    unsigned curr_node = 0;
    //iterate over nodes in a subtree
    while (true){
        unsigned feature_id    = node_features[curr_node]; 
        float node_value       = node_values[curr_node];
        unsigned is_tree_leaf  = node_is_leaf[curr_node];
        // if node is leaf, then the prediction is over, we return the predicted value in node_value (in a tree leaf, node_value holds the predicted result)
        if (is_tree_leaf==1)
            return node_value;
        // if node is not leaf, we need two comparisons to decide if we keep traverse 
        bool go_left = row[feature_id] <= node_value;
        if (go_left)
            curr_node = edge_list[node_list[curr_node]]; 
        // go to right child in subtree
        else
            curr_node = edge_list[node_list[curr_node]+1]; 
    }
}




//Alternative way to traverse a node
//                    if (go_left){
//                        if(not_subtree_bottom){
//                          curr_node = curr_node*2 + 1;
//                        }else{
//                          unsigned leaf_idx = curr_node - subtree_leaf_idx_boundry;
//                          curr_subtree_idx = subtree_idx_to_subtree[2*leaf_idx];
//                          break;
//                        }
//                    // go to right child in subtree
//                    } else {
//                        if(not_subtree_bottom){
//                          curr_node = curr_node*2 + 2;
//                        }else{
//                          unsigned leaf_idx = curr_node - subtree_leaf_idx_boundry;
//                          curr_subtree_idx = subtree_idx_to_subtree[2*leaf_idx+1];
//                          break;
//                        }
//                    }
